#include "hip/hip_runtime.h"

// GPU kernel
__global__ void summation_kernel_commom(int data_size, results * data_out)
{
	int num_threads = gridDim.x * blockDim.x;
	int thread_data_size = data_size / num_threads;
	int thread_absolute_id = blockIdx.x * blockDim.x + threadIdx.x;

        float result = 0;
        for (int i = thread_absolute_id*thread_data_size; i<thread_absolute_id*thread_data_size+thread_data_size; i++) {
                result = result + (float)((((i%2)-1)+(i%2)))*-1. / (float)(i+1);
        }
	data_out[thread_absolute_id].sum = result;

}


__global__ void summation_kernel_interleaved(int data_size, results * data_out)
{

	int num_threads = gridDim.x * blockDim.x;
	int thread_data_step = data_size / num_threads;
	int thread_absolute_id = blockIdx.x * blockDim.x + threadIdx.x;


        float result = 0;
	int i;
	for(int j = 0; j < thread_data_step; j++){
        	i = j * num_threads + thread_absolute_id;
                result = result + (float)((((i%2)-1)+(i%2)))*-1. / (float)(i+1);
        }
	data_out[thread_absolute_id].sum = result;
}


__global__ void summation_kernel_per_block(int data_size, results * data_out)
{
	extern __shared__ float sum_threads[];

	int num_threads = gridDim.x * blockDim.x;
	int thread_data_size = data_size / num_threads;
	int thread_absolute_id = blockIdx.x * blockDim.x + threadIdx.x;

        float result = 0;
        for (int i = thread_absolute_id*thread_data_size; i<thread_absolute_id*thread_data_size+thread_data_size; i++) {
                result = result + (float)((((i%2)-1)+(i%2)))*-1. / (float)(i+1);
        }
	sum_threads[threadIdx.x] = result;

	__syncthreads();
	
        for (int i = 1 ; i<blockDim.x ; i = i << 1 ) {
                if (threadIdx.x%(i << 1) == 0) {
                        sum_threads[threadIdx.x] += sum_threads[threadIdx.x+i];
                }
		__syncthreads();
        }

	data_out[blockIdx.x].sum = sum_threads[0];

}
