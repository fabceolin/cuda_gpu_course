
#include "utils.h"
#include <stdlib.h>

#include "life_kernel.cu"


int main(int argc, char ** argv)
{
    // Definition of parameters
    int domain_x = 128;// Multiple of threads_per_block * cell_per_word
    int domain_y = 128;

    int cells_per_word = 1;

    int steps = 2;


    int blocks_y_step = 4;
    int threads_per_block = 128 * blocks_y_step;
    int blocks_x = blocks_y_step * domain_x / (threads_per_block * cells_per_word);
    int blocks_y = domain_y / blocks_y_step;

    dim3  grid(blocks_x  , blocks_y );// CUDA grid dimensions
    dim3  threads(threads_per_block);// CUDA block dimensions

    // Allocation of arrays
    int * domain_gpu[2] = {NULL, NULL};

    size_t pitch;
    CUDA_SAFE_CALL(hipMallocPitch((void**)&domain_gpu[0], &pitch,
                domain_x / cells_per_word * sizeof(int),
                domain_y));
    CUDA_SAFE_CALL(hipMallocPitch((void**)&domain_gpu[1], &pitch,
                domain_x / cells_per_word * sizeof(int),
                domain_y));

    // Arrays of dimensions pitch * domain.y
    init_kernel<<< grid, threads, 0 >>>(domain_gpu[0], pitch);

    // Timer initialization
    hipEvent_t start, stop;
    CUDA_SAFE_CALL(hipEventCreate(&start));
    CUDA_SAFE_CALL(hipEventCreate(&stop));

    // Start timer
    CUDA_SAFE_CALL(hipEventRecord(start, 0));

    // Kernel execution
    int shared_mem_size = domain_x * (blocks_y_step+2) * sizeof(int) ;
    printf("%d %d %d \n",blocks_x, blocks_y, shared_mem_size);
    for(int i = 0; i < steps; i++) {
        life_kernel<<< grid, threads, shared_mem_size >>>(domain_gpu[i%2], domain_gpu[(i+1)%2], domain_x, domain_y, pitch, blocks_y_step);
    }

    // Stop timer
    CUDA_SAFE_CALL(hipEventRecord(stop, 0));
    CUDA_SAFE_CALL(hipEventSynchronize(stop));

    float elapsedTime;
    CUDA_SAFE_CALL(hipEventElapsedTime(&elapsedTime, start, stop));	// In ms
    printf("GPU time: %f ms\n", elapsedTime);

    CUDA_SAFE_CALL(hipEventDestroy(start));
    CUDA_SAFE_CALL(hipEventDestroy(stop));

    // Get results back
    int * domain_cpu = (int*)malloc(pitch * domain_y);
    CUDA_SAFE_CALL(hipMemcpy(domain_cpu, domain_gpu[steps%2], pitch * domain_y, hipMemcpyDeviceToHost));

    CUDA_SAFE_CALL(hipFree(domain_gpu[0]));
    CUDA_SAFE_CALL(hipFree(domain_gpu[1]));


    // Count colors
    int red = 0;
    int blue = 0;
    for(int y = 0; y < domain_y; y++)
    {
        for(int x = 0; x < domain_x; x++)
        {
            int cell = domain_cpu[y * pitch/sizeof(int) + x];
            printf("%u", cell);
            if(cell == 1) {
                red++;
            }
            else if(cell == 2) {
                blue++;
            }
        }
        printf("\n");
    }

    printf("Red/Blue cells: %d/%d\n", red, blue);

    free(domain_cpu);

    return 0;
}

