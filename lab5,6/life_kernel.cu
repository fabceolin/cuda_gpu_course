#include "hip/hip_runtime.h"

__global__ void init_kernel(int * domain, int pitch, int block_y_step)
{
                          /* 512 / 4 */

    int tx = threadIdx.x % blockDim.x;
    int ty = (blockIdx.y * blockDim.y) + threadIdx.y;

    int value = tx % 3;
    switch (value) {
        case(0):
            domain[ tx + ty * blockDim.x] = 1;
            break;
        case(1):
            domain[ tx + ty * blockDim.x] = 0;
            break;
        case(2):
            domain[ tx + ty * blockDim.x] = 2;
            break;
    }

}

// Reads a cell at (x+dx, y+dy)
__device__ int read_cell(int * source_domain, int x, int y, int dx, int dy,
    int domain_x, int domain_y, int pitch)
{
    x = (unsigned)  (x + dx) % domain_x;// Wrap around
    y = (unsigned)  (y + dy) % domain_y;
    return source_domain[y * (pitch / sizeof(int)) + x];
}

__device__ void write_cell(int * dest_domain, int x, int y, int dx, int dy,
    int domain_x, int domain_y, int pitch, int value)
{
    x = (unsigned)(x + dx) % domain_x; // Wrap around
    y = (unsigned)(y + dy) % domain_y;
    dest_domain[y * (pitch / sizeof(int)) + x] = value;
}

// Compute kernel
__global__ void life_kernel(int * source_domain, int * dest_domain, int domain_x, int domain_y, int pitch)
{
    extern __shared__ int shared_data[];

    int init_tx = threadIdx.x * CELLS_PER_THREADS;
            /* 0-511 */
            /*       0-127           */

/*  
    global memory 
    X  --------->                                                                                                                 Y
  00210210210210210210210210210210210210210210210210210210210210210210210210210020100210211002210210210210210210020100020002210210 |
  01000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000001 |
  11000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000001 V
  11000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000001
  */

    int ty = blockIdx.y * blockDim.y + (threadIdx.y);

    int init_shared_tx = init_tx;
    int shared_ty = ty % blockDim.y + 1;

    // load shared;
    /*
 
                                                                                                                                   127
   0                                                                                                                              /
       // Shared memory                                                                                                           |
0  00210210210210210210210210210210210210210210210210210210210210210210210210210020100210211002210210210210210210020100020002210210

   X(shared_x=0, shared_y= 1)  --------->                                                                                         Y
1  00210210210210210210210210210210210210210210210210210210210210210210210210210020100210211002210210210210210210020100020002210210 |
2  01000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000001 |
3  11000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000001 V
4  11000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000001

5  11000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000000001
       */
    for (int i=0 ; i<CELLS_PER_THREADS; i++ ) {
        shared_data[init_shared_tx + i + (shared_ty)*blockDim.x*CELLS_PER_THREADS] = read_cell(source_domain, init_tx, ty, i, 0, domain_x, domain_y, pitch);
        if (shared_ty == 1) {
            shared_data[init_shared_tx + i + (shared_ty-1)*blockDim.x*CELLS_PER_THREADS] = read_cell(source_domain, init_tx, ty, i, -1, domain_x, domain_y, pitch);
        }
        if (shared_ty == 4) {
            shared_data[init_shared_tx + i + (shared_ty+1)*blockDim.x*CELLS_PER_THREADS] = read_cell(source_domain, init_tx, ty, i, 1, domain_x, domain_y, pitch);
        }
    }
    __syncthreads();
    if ( (threadIdx.x == 0) && (threadIdx.y==0) && (blockIdx.y==0 )) {
        int i;
        for (i=0;i<768;i++) {
            write_cell(dest_domain, i%(blockDim.x*CELLS_PER_THREADS), i/(blockDim.x*CELLS_PER_THREADS), 0,0,domain_x,domain_y,pitch,(shared_data[i])%10);
        }
    }
    return;
#if 0
    __syncthreads();
    // Read cell
    int myself = shared_data[shared_tx + (shared_ty)*blockDim.x];


    // TODO: Read the 8 neighbors and count number of blue and red
    int blue=0;
    int red=0;
    int adjacent_count=0;
    for (int i=0; i<9;i++) {
        if (i==4) /* itself */ {
            continue;
        }
        int x = i % 3 - 1;
        int y = (int) (i / 3) - 1;
        int near = shared_data[(((x+shared_tx+blockDim.x)%blockDim.x) + ((shared_ty+y)*blockDim.x))];
        switch (near) {
            case (1):
                red++;
                break;
            case (2):
                blue++;
                break;
            default:
                break;
        }
        adjacent_count = adjacent_count + (!((i+1)%2) && near);
    }

    int total_near = blue+red;
    int new_value = myself;
    // rules
    if ((total_near)>3) {
        new_value = 0;
    }

    if (adjacent_count==1) {
        new_value = 0;
    }

    if ((total_near)==3 && (myself==0)) {
        new_value = 1 << ((blue & 0x02) >> 1);
    }

    write_cell(dest_domain, tx, ty, 0,0,domain_x,domain_y,pitch,new_value);
    return;
#endif
}

