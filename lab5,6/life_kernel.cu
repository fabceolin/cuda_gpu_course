
#include <hip/hip_runtime.h>

__global__ void init_kernel(int * domain, int pitch)
{
    domain[blockIdx.y * pitch / sizeof(int) + blockIdx.x * blockDim.x + threadIdx.x]
        = (1664525ul * (blockIdx.x + threadIdx.y + threadIdx.x) + 1013904223ul) % 3;
}

// Reads a cell at (x+dx, y+dy)
__device__ int read_cell(int * source_domain, int x, int y, int dx, int dy,
    int domain_x, int domain_y, int pitch)
{
    x = (x + dx) % domain_x;	// Wrap around
    y = (y + dy) % domain_y;
    return source_domain[y * (pitch / sizeof(int)) + x];
}

__device__ void write_cell(int * dest_domain, int x, int y, int dx, int dy,
    int domain_x, int domain_y, int pitch, int value)
{
    x = (x + dx) % domain_x;	// Wrap around
    y = (y + dy) % domain_y;
    dest_domain[y * (pitch / sizeof(int)) + x] = value;
}

// Compute kernel
__global__ void life_kernel(int * source_domain, int * dest_domain, int domain_x, int domain_y, int pitch, int block_y_step)
{
    extern __shared__ int shared_data[];

    int tx = blockIdx.x * blockDim.x + threadIdx.x % blockDim.x ;
    int ty = blockIdx.y * block_y_step + (int)(threadIdx.x / blockDim.x);  // Conta com o stem;

    // load shared;
    shared_data[tx * (1+ty) ] = read_cell(source_domain, tx, ty, 0, 0,
                       domain_x, domain_y, pitch);

    if (threadIdx.y == 0 ) {
        shared_data[tx * (ty)] = read_cell(source_domain, tx, ty, 0, -1,
                       domain_x, domain_y, pitch);
    }

    if (threadIdx.y == 3 ) {
        shared_data[tx * (ty+block_y_step)] = read_cell(source_domain, tx, ty, 0, 1,
                       domain_x, domain_y, pitch);
    }

    __syncthreads();

    // Read cell
    int myself = shared_data[tx * ty];


    // TODO: Read the 8 neighbors and count number of blue and red
    int blue=0;
    int red=0;
    int adjacent_count=0;
    for (int i=0; i<9;i++) {
        if (i==4) /* itself */ {
            continue;
        }
        int x = i % 3 - 1;
        int y = (int) (i / 3) - 1;
        int near = shared_data[(x+tx)*(y+ty)];
        switch (near) {
            case (1):
                red++;
                break;
            case (2):
                blue++;
                break;
            default:
                break;
        }
        if ( (i+1)%2==0) {
            if (near>0) {
                adjacent_count++;
            }
        }
    }

    int total_near = blue+red;
    int new_value = myself;
    // rules
    if ((total_near)>3) {
        new_value = 0;
    }

    if (adjacent_count==1) {
        new_value = 0;
    }
    if ((total_near)==3 && (myself==0)) {
         if (blue>red) {
             new_value=2;
         }
         else {
             new_value=1;
         }
    }

    write_cell(dest_domain, tx, ty, 0,0,domain_x,domain_y,pitch,new_value);
    return;

}

